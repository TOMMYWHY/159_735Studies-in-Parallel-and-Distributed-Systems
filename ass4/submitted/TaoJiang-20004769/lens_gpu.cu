#include "hip/hip_runtime.h"
/* 
   159735 Parallel Programming

   Startup program for sequential implementation of simulation by ray
   tracing of gravitational lensing.
 */
 #include <ctime>

 #include <iostream>
 #include <string>
 
 #include <cmath>
 #include <hip/hip_runtime.h>
 
 #include "lenses.h"
 #include "arrayff.hxx"
 #include <fstream>
 
 using namespace std;
 const float WL  = 2.0;
 const float XL1 = -WL;
 const float XL2 =  WL;
 const float YL1 = -WL;
 const float YL2 =  WL;
 
 // kernel
 __global__ void cuda_shoot(float* lensim, int xsize, int ysize, float lens_scale, float* xlens, float* ylens, float* eps, int nlenses)
 {
   const float xsrc = 0.0;      
   const float ysrc = 0.0;
   const float rsrc = 0.1;     
   const float rsrc2 = rsrc * rsrc;
   const float ldc  = 0.5;    
 
   int index = threadIdx.x + blockIdx.x * blockDim.x;
   if (index >= xsize*ysize)
     return;
 
   int iy = index / xsize;
   int ix = index % xsize;
 
   float yl = -2.0 + iy * lens_scale;  
   float xl = -2.0 + ix * lens_scale;  
   float xs = xl;
   float ys = yl;
 
   for (int p = 0; p < nlenses; ++p) {
     float dx = xl - xlens[p];
     float dy = yl - ylens[p];
     float dr = dx * dx + dy * dy;
     xs -= eps[p] * dx / dr;
     ys -= eps[p] * dy / dr;
   }
 
   float xd = xs - xsrc;
   float yd = ys - ysrc;
   float sep2 = xd * xd + yd * yd;
   if (sep2 < rsrc2) {
     float mu = sqrt(1 - sep2 / rsrc2);
     lensim[index] = 1.0 - ldc * (1 - mu);
   }
 }

 double diffclock(clock_t clock1,clock_t clock2)
 {
   double diffticks = clock1 - clock2;
   double diffms = (diffticks * 1000) / CLOCKS_PER_SEC;
   return diffms; // Time difference in milliseconds
 }
 
 double seq(int len_num);

 int main(int argc, char* argv[]) 
 {
     const int len_num = atoi(argv[1]);

    //  cout << "-------------------------"<<endl;
     float* xlens;
     float* ylens;
     float* eps;
 //    const int nlenses = set_example_3(&xlens, &ylens, &eps);
     const int nlenses = set_example_n(len_num, &xlens, &ylens, &eps);
 
    //  std::cout << "# Simulating " << nlenses << " lens system" << std::endl;
 
   const float lens_scale = 0.0025;
   const int npixx = static_cast<int>(floor((XL2 - XL1) / lens_scale)) + 1;
   const int npixy = static_cast<int>(floor((YL2 - YL1) / lens_scale)) + 1;
  //  std::cout << "# Building " << npixx << "X" << npixy << " lens image" << std::endl;
 
   Array<float, 2> lensim(npixy, npixx);
  //  cout << "lensim.ntotal: "<<lensim.ntotal <<endl;
   //copy data to GPU space

   float* d_xlens  ,*d_ylens  ,*d_eps    ,*d_lensim;

   hipMalloc(&d_xlens, nlenses*sizeof(float));
   hipMalloc(&d_ylens, nlenses*sizeof(float));
   hipMalloc(&d_eps, nlenses*sizeof(float));
   hipMalloc(&d_lensim, lensim.ntotal*sizeof(float));
   
  //  // Copy vectors from host to device memory
   hipMemcpy(d_xlens, xlens, nlenses*sizeof(float), hipMemcpyHostToDevice);
   hipMemcpy(d_ylens, ylens, nlenses*sizeof(float), hipMemcpyHostToDevice);
   hipMemcpy(d_eps, eps, nlenses*sizeof(float), hipMemcpyHostToDevice);
   hipMemcpy(d_lensim, lensim.buffer,lensim.ntotal*sizeof(float), hipMemcpyHostToDevice);
 
     
 
   clock_t tstart = clock();
 
   int threadsPerBlock = 256;
  int blocksPerGrid = lensim.ntotal / threadsPerBlock + 1;
 
  //  int blocksPerGrid = (lensim.ntotal+ threadsPerBlock-1)/ threadsPerBlock ;
   std::cout << "Launching a grid of " << blocksPerGrid
         << " "
         << threadsPerBlock * blocksPerGrid
         << " threads" << std::endl;
   cuda_shoot<<<blocksPerGrid, threadsPerBlock>>>(d_lensim, npixx, npixy, lens_scale, d_xlens, d_ylens, d_eps, nlenses);
   
   clock_t tend = clock();
   double tms = diffclock(tend, tstart);
   std::cout << "# Time cost: " << tms << " ms;--- " << tms/1000 << "s" << std::endl;
 
   //copy the lens image to host
   hipMemcpy(lensim.buffer, d_lensim, sizeof(float)*lensim.ntotal, hipMemcpyDeviceToHost);
 
   dump_array<float, 2>(lensim, "lens_gpu.fit");
  /*----------------------*/
   double seq_cost= seq( len_num);


   ofstream file("report.txt",ios::app);
    if (file.is_open()){
        file <<"nlenses:"<< to_string(nlenses)<<
        "; seq:"<<to_string(seq_cost);
        file << "; cuda use " <<to_string(tms) << "\n";
        file.close();
    }
 
   hipFree(d_xlens);
   hipFree(d_ylens);
   hipFree(d_eps);
   hipFree(d_lensim);
 
   delete[] xlens;
   delete[] ylens;
   delete[] eps;
 }
 
/***************************************************/
 double seq(int len_num){
  float* xlens;
  float* ylens;
  float* eps;
  const int nlenses = set_example_n(len_num, &xlens, &ylens, &eps);
  std::cout << "# Simulating " << nlenses << " lens system" << std::endl;

  const float rsrc = 0.1;      // radius
  const float ldc  = 0.5;      // limb darkening coefficient
  const float xsrc = 0.0;      // x and y centre on the map
  const float ysrc = 0.0;

  const float lens_scale = 0.0025;

  const int npixx = static_cast<int>(floor((XL2 - XL1) / lens_scale)) + 1;
  const int npixy = static_cast<int>(floor((YL2 - YL1) / lens_scale)) + 1;
  Array<float, 2> lensim(npixy, npixx);

  clock_t tstart = clock();

  const float rsrc2 = rsrc * rsrc;
  float xl, yl, xs, ys, sep2, mu;
  float xd, yd;
  int numuse = 0;
  for (int iy = 0; iy < npixy; ++iy)
      for (int ix = 0; ix < npixx; ++ix) {
          yl = YL1 + iy * lens_scale;
          xl = XL1 + ix * lens_scale;

          shoot(xs, ys, xl, yl, xlens, ylens, eps, nlenses);

          xd = xs - xsrc;
          yd = ys - ysrc;
          sep2 = xd * xd + yd * yd;
          if (sep2 < rsrc2) {
              mu = sqrt(1 - sep2 / rsrc2);
              lensim(iy, ix) = 1.0 - ldc * (1 - mu);
          }
      }

  clock_t tend = clock();
  double tms = diffclock(tend, tstart);
  std::cout << "# seq Time cost: " << tms << " ms;--- " << tms/1000 << "s" << std::endl;

  // dump_array<float, 2>(lensim, "lens_seq.fit");
  
  return tms;
  
  delete[] xlens;
  delete[] ylens;
  delete[] eps;
 }
 
